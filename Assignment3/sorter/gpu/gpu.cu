#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstddef>
#include <vector>

#include "device_buffer.h"
#include "gpu.h"

// Returns the largest power of two that is less than or equal to the input
__host__ __device__ size_t largest_power_of_two(size_t n) {
    size_t power = 1;
    while (power * 2 <= n) {
        power *= 2;
    }
    return power;
}

// Performs a bitonic sort on consecutive blocks of elements
// Each block of elements is sorted by a single thread block
// The direction of the sort for a block is determined by the direction bit of the block index
__global__ void bitonic_sort_blockwise(device_buffer_t<element_t> data, size_t block_direction_bit_pos) {
    assert(blockDim.x == GPU_BLOCK_SIZE / 2);

    // Find the start index of the block
    const size_t block_start = blockIdx.x * blockDim.x;
    // And the ID of this thread within the block
    const size_t worker_id = threadIdx.x;

    // Calculate the direction for this block
    // The masked bit of the thread block index determines the direction of the sort
    bool block_ascending = (blockIdx.x & (1 << block_direction_bit_pos)) == 0;

    // Load the block of elements into shared memory
    __shared__ element_t shared_data[GPU_BLOCK_SIZE / 2];
    shared_data[worker_id] = data[block_start + worker_id];
    __syncthreads();

    // Run an outer loop to sort a range of elements at a time
    // For each iteration, sort_range number of elements are sorted
    for (size_t sort_range = 2; sort_range <= GPU_BLOCK_SIZE / 2; sort_range *= 2) {
        // Find the sort group we are working on
        const size_t sort_group_num = worker_id / sort_range;
        // And the ID of this thread within the sort group
        size_t sort_group_id = worker_id % sort_range;

        // Every other group of elements is sorted in the opposite direction
        const bool ascending = sort_group_num % 2 == 0 ? block_ascending : !block_ascending;

        // Run an inner loop to compare and swap elements
        for (size_t compare_range = sort_range; compare_range > 1; compare_range /= 2) {
            // Find the comparison group we are working on
            const size_t compare_group_num = sort_group_id / compare_range;
            // And the ID of this thread within the comparison group
            const size_t compare_group_id = sort_group_id % compare_range;

            // Calculate the index of the element this thread is assigned to
            const size_t element_id = sort_group_num * sort_range + compare_group_num * compare_range + compare_group_id;
            // And the index of the element to compare with using bitwise XOR
            const size_t compare_id = element_id ^ (compare_range / 2);

            // Make sure the compare_id is not behind the element_id
            if (element_id < compare_id) {
                // Load the elements to compare
                const element_t element = shared_data[element_id];
                const element_t compare = shared_data[compare_id];

                // Compare and swap the elements if necessary
                if ((element > compare) == ascending) {
                    shared_data[element_id] = compare;
                    shared_data[compare_id] = element;
                }
            }

            __syncthreads();
        }

        __syncthreads();
    }

    // Store the sorted block of elements back into global memory
    data[block_start + worker_id] = shared_data[worker_id];
}

// __host__ void bitonic_sort_driver(device_buffer_t<element_t> &data, size_t range_start, size_t num_blocks) {
//     // If there is only one block of elements, launch the kernel to sort the block
//     if (num_blocks == 1) {
//         bitonic_sort_kernel<<<1, GPU_BLOCK_SIZE>>>(data, range_start, 0);
//         return;
//     }

//     // For larger numbers of blocks, start with a stride of 2 and go up to the number of blocks
//     for (size_t stride = 2; stride <= num_blocks; stride *= 2) {
//         // At each stride, perform a bitonic sort on the blocks of elements
//         bitonic_sort_kernel<<<num_blocks, GPU_BLOCK_SIZE>>>(data, range_start, stride - 1);

//         // Wait for the kernel to finish
//         CUDA_CHECK(hipPeekAtLastError());
//         CUDA_CHECK(hipDeviceSynchronize());

//         // Then launch a kernel to merge the blocks of elements
//         bitonic_merge_kernel<<<num_blocks, GPU_BLOCK_SIZE>>>(data, range_start, stride);

//     }
// }

// Merges the sorted blocks of data on the CPU recursively using OpenMP tasks
void do_final_merge(std::vector<element_t> &data, std::vector<size_t> &sorted_block_starts, size_t start, size_t end) {
    // If there is only one block of data, there is nothing to merge
    if (end - start <= 1) {
        return;
    }

    // Otherwise, split the blocks of data in half and merge them recursively
    size_t mid = (end - start) / 2;

    #pragma omp task shared(data, sorted_block_starts)
    do_final_merge(data, sorted_block_starts, start, start + mid);
    #pragma omp task shared(data, sorted_block_starts)
    do_final_merge(data, sorted_block_starts, start + mid, end);
    #pragma omp taskwait

    std::inplace_merge(
        data.begin() + sorted_block_starts[start],
        data.begin() + sorted_block_starts[start + mid],
        data.begin() + sorted_block_starts[end]);
}

__host__ void sort_range_gpu(std::vector<element_t> &data, size_t range_start, size_t range_end) {
    size_t num_elements_total = range_end - range_start;

    // Exit early if there are no elements to sort
    if (num_elements_total == 0) {
        return;
    }

    // Check if there are indeed any CUDA devices
    int device_count;
    CUDA_CHECK(hipGetDeviceCount(&device_count));
    if (device_count == 0) {
        std::cout << "No CUDA Devices found!" << std::endl;
        std::exit(1);
    }

    // Find the largest power-of-2 number of elements that can fit on 80% of the GPU memory
    size_t total_gpu_memory;
    CUDA_CHECK(hipMemGetInfo(nullptr, &total_gpu_memory));

    size_t safe_gpu_memory = total_gpu_memory * 0.8;
    size_t max_elements = largest_power_of_two(safe_gpu_memory / sizeof(element_t));

    // Allocate memory on the GPU for the blocks of data assigned to the GPU
    device_buffer_t<element_t> device_data;
    // Also keep track of where the sorted blocks start
    std::vector<size_t> sorted_block_starts;

    size_t buffer_size = std::min(max_elements, num_elements_total);
    for (size_t i = range_start; i < range_end; i += buffer_size) {
        // If there are fewer elements than the buffer size, reduce the buffer size to the next power of two
        if (i + buffer_size > range_end) {
            buffer_size = largest_power_of_two(range_end - i);
        }

        // Reallocate the device buffer if the number of elements is different
        if (device_data.size() != buffer_size) {
            device_data.reallocate(buffer_size);
        }

        // Copy the blocks of data to the GPU
        device_data.copy_to_device(&data[i], buffer_size);

        // Launch the kernel to sort the blocks of data
        std::cout << "GPU Block Sort Start" << std::endl;
        bitonic_sort_blockwise<<<buffer_size / (GPU_BLOCK_SIZE / 2), GPU_BLOCK_SIZE / 2>>>(device_data, 0);
        // Wait for the kernel to finish
        CUDA_CHECK(hipPeekAtLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        std::cout << "GPU Block Sort End" << std::endl;

        // Copy the sorted data back to the CPU
        device_data.copy_from_device(&data[i], buffer_size);
        // And mark the start of the sorted block
        sorted_block_starts.push_back(i);
    }

    // Do the final merge on the CPU
    do_final_merge(data, sorted_block_starts, 0, sorted_block_starts.size());
}
